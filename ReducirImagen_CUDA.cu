#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <pthread.h>
#include <omp.h>
#include <sys/time.h>
#include <fstream>
#include <string> 

using namespace std;
using namespace cv;

int BLOCKSPERGRID  = 32;
int NUMTHREADS = 128;

int **outR;
int **outG;
int **outB;
int **imgR;
int **imgG;
int **imgB;
int numeroColumnasImg = 0;

void matriz4x4Amatriz2x2(int imgR[4][4], int imgG[4][4], int imgB[4][4], int outR[2][2], int outG[2][2], int outB[2][2])
{ // recibo una matríz cuadrada con numero par de filas (nxn)
    /*cuadrantes
    --- ---
    |1 | 2|
    --- ---
    |3 | 4|
    --- ---*/
    int n = sizeof(*imgR) / sizeof(*imgR[0]);
    int centroMatriz = (n / 2);
    int cuadrante1R = 0, cuadrante2R = 0, cuadrante3R = 0, cuadrante4R = 0;
    int cuadrante1G = 0, cuadrante2G = 0, cuadrante3G = 0, cuadrante4G = 0;
    int cuadrante1B = 0, cuadrante2B = 0, cuadrante3B = 0, cuadrante4B = 0;

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (i < centroMatriz)
            {
                if (j < centroMatriz)
                {
                    //cuadrante 1
                    cuadrante1R += imgR[i][j];
                    cuadrante1G += imgG[i][j];
                    cuadrante1B += imgB[i][j];
                }
                else
                {
                    //cuadrante 2
                    cuadrante2R += imgR[i][j];
                    cuadrante2G += imgG[i][j];
                    cuadrante2B += imgB[i][j];
                }
            }
            else
            {
                if (j < centroMatriz)
                {
                    //cuadrante 3
                    cuadrante3R += imgR[i][j];
                    cuadrante3G += imgG[i][j];
                    cuadrante3B += imgB[i][j];
                }
                else
                {
                    //cuadrante 4
                    cuadrante4R += imgR[i][j];
                    cuadrante4G += imgG[i][j];
                    cuadrante4B += imgB[i][j];
                }
            }
        }
    }

    int divisor = centroMatriz * centroMatriz;

    outR[0][0] = cuadrante1R / divisor;
    outR[0][1] = cuadrante2R / divisor;
    outR[1][0] = cuadrante3R / divisor;
    outR[1][1] = cuadrante4R / divisor;

    outG[0][0] = cuadrante1G / divisor;
    outG[0][1] = cuadrante2G / divisor;
    outG[1][0] = cuadrante3G / divisor;
    outG[1][1] = cuadrante4G / divisor;

    outB[0][0] = cuadrante1B / divisor;
    outB[0][1] = cuadrante2B / divisor;
    outB[1][0] = cuadrante3B / divisor;
    outB[1][1] = cuadrante4B / divisor;
}

void algoritmo2Para4K(int imgR[8][8], int imgG[8][8], int imgB[8][8], int outR[2][2], int outG[2][2], int outB[2][2])
{ // recibo una matríz cuadrada con numero par de filas (nxn)
    /*cuadrantes
    --- ---
    |1 | 2|
    --- ---
    |3 | 4|
    --- ---*/
    int n = sizeof(*imgR) / sizeof(*imgR[0]);
    int centroMatriz = (n / 2);
    int cuadrante1R = 0, cuadrante2R = 0, cuadrante3R = 0, cuadrante4R = 0;
    int cuadrante1G = 0, cuadrante2G = 0, cuadrante3G = 0, cuadrante4G = 0;
    int cuadrante1B = 0, cuadrante2B = 0, cuadrante3B = 0, cuadrante4B = 0;

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (i < centroMatriz)
            {
                if (j < centroMatriz)
                {
                    //cuadrante 1
                    cuadrante1R += imgR[i][j];
                    cuadrante1G += imgG[i][j];
                    cuadrante1B += imgB[i][j];
                }
                else
                {
                    //cuadrante 2
                    cuadrante2R += imgR[i][j];
                    cuadrante2G += imgG[i][j];
                    cuadrante2B += imgB[i][j];
                }
            }
            else
            {
                if (j < centroMatriz)
                {
                    //cuadrante 3
                    cuadrante3R += imgR[i][j];
                    cuadrante3G += imgG[i][j];
                    cuadrante3B += imgB[i][j];
                }
                else
                {
                    //cuadrante 4
                    cuadrante4R += imgR[i][j];
                    cuadrante4G += imgG[i][j];
                    cuadrante4B += imgB[i][j];
                }
            }
        }
    }

    int divisor = centroMatriz * centroMatriz;

    outR[0][0] = cuadrante1R / divisor;
    outR[0][1] = cuadrante2R / divisor;
    outR[1][0] = cuadrante3R / divisor;
    outR[1][1] = cuadrante4R / divisor;

    outG[0][0] = cuadrante1G / divisor;
    outG[0][1] = cuadrante2G / divisor;
    outG[1][0] = cuadrante3G / divisor;
    outG[1][1] = cuadrante4G / divisor;

    outB[0][0] = cuadrante1B / divisor;
    outB[0][1] = cuadrante2B / divisor;
    outB[1][0] = cuadrante3B / divisor;
    outB[1][1] = cuadrante4B / divisor;
}

void algoritmo2Para1080p(int imgR[8][8], int imgG[8][8], int imgB[8][8], int outR[4][4], int outG[4][4], int outB[4][4])
{ // recibo matriz de 8x8 y tamaño de matriz resultante (nxn)
    int subMatriz1R[4][4];
    int subMatriz2R[4][4];
    int subMatriz3R[4][4];
    int subMatriz4R[4][4];

    int subMatriz1G[4][4];
    int subMatriz2G[4][4];
    int subMatriz3G[4][4];
    int subMatriz4G[4][4];

    int subMatriz1B[4][4];
    int subMatriz2B[4][4];
    int subMatriz3B[4][4];
    int subMatriz4B[4][4];
    int centroMatriz = 4;

    for (int i = 0; i < 8; i++)
    {
        for (int j = 0; j < 8; j++)
        {
            if (i < centroMatriz)
            {
                if (j < centroMatriz)
                {
                    //cuadrante 1
                    subMatriz1R[i][j] = imgR[i][j];
                    subMatriz1G[i][j] = imgG[i][j];
                    subMatriz1B[i][j] = imgB[i][j];
                }
                else
                {
                    //cuadrante 2
                    subMatriz2R[i][j - 4] = imgR[i][j];
                    subMatriz2G[i][j - 4] = imgG[i][j];
                    subMatriz2B[i][j - 4] = imgB[i][j];
                }
            }
            else
            {
                if (j < centroMatriz)
                {
                    //cuadrante 3
                    subMatriz3R[i - 4][j] = imgR[i][j];
                    subMatriz3G[i - 4][j] = imgG[i][j];
                    subMatriz3B[i - 4][j] = imgB[i][j];
                }
                else
                {
                    //cuadrante 4
                    subMatriz4R[i - 4][j - 4] = imgR[i][j];
                    subMatriz4G[i - 4][j - 4] = imgG[i][j];
                    subMatriz4B[i - 4][j - 4] = imgB[i][j];
                }
            }
        }
    }

    int matrizCuadrante1R[2][2];
    int matrizCuadrante2R[2][2];
    int matrizCuadrante3R[2][2];
    int matrizCuadrante4R[2][2];

    int matrizCuadrante1G[2][2];
    int matrizCuadrante2G[2][2];
    int matrizCuadrante3G[2][2];
    int matrizCuadrante4G[2][2];

    int matrizCuadrante1B[2][2];
    int matrizCuadrante2B[2][2];
    int matrizCuadrante3B[2][2];
    int matrizCuadrante4B[2][2];

    matriz4x4Amatriz2x2(subMatriz1R, subMatriz1G, subMatriz1B, matrizCuadrante1R, matrizCuadrante1G, matrizCuadrante1B);
    matriz4x4Amatriz2x2(subMatriz2R, subMatriz2G, subMatriz2B, matrizCuadrante2R, matrizCuadrante2G, matrizCuadrante2B);
    matriz4x4Amatriz2x2(subMatriz3R, subMatriz3G, subMatriz3B, matrizCuadrante3R, matrizCuadrante3G, matrizCuadrante3B);
    matriz4x4Amatriz2x2(subMatriz4R, subMatriz4G, subMatriz4B, matrizCuadrante4R, matrizCuadrante4G, matrizCuadrante4B);

    for (int i = 0; i < 2; i++)
    {
        for (int j = 0; j < 2; j++)
        {
            outR[i][j] = matrizCuadrante1R[i][j];
            outR[i][j + 2] = matrizCuadrante2R[i][j];
            outR[i + 2][j] = matrizCuadrante3R[i][j];
            outR[i + 2][j + 2] = matrizCuadrante4R[i][j];

            outG[i][j] = matrizCuadrante1G[i][j];
            outG[i][j + 2] = matrizCuadrante2G[i][j];
            outG[i + 2][j] = matrizCuadrante3G[i][j];
            outG[i + 2][j + 2] = matrizCuadrante4G[i][j];

            outB[i][j] = matrizCuadrante1B[i][j];
            outB[i][j + 2] = matrizCuadrante2B[i][j];
            outB[i + 2][j] = matrizCuadrante3B[i][j];
            outB[i + 2][j + 2] = matrizCuadrante4B[i][j];
        }
    }
}

Mat cambiarTamanoImagen(Mat img, int row, int nuevoNcolumnas)
{
    Mat imgAux(row, nuevoNcolumnas, CV_8UC3);

    for (int i = 0; i < imgAux.rows; i++)
    {
        for (int j = 0; j < imgAux.cols; j++)
        {
            imgAux.at<cv::Vec3b>(i, j)[0] = img.at<cv::Vec3b>(i, j)[0];
            imgAux.at<cv::Vec3b>(i, j)[1] = img.at<cv::Vec3b>(i, j)[1];
            imgAux.at<cv::Vec3b>(i, j)[2] = img.at<cv::Vec3b>(i, j)[2];
        }
    }
    return imgAux;
}

__global__ void reducirMatriz3x3a2x2(int imgR[3][3], int imgG[3][3], int imgB[3][3], int outR[2][2], int outG[2][2], int outB[2][2])
{
    double R[3][2];
    double G[3][2];
    double B[3][2];

    for (int k = 0; k < 3; k++)
    {
        for (int l = 0; l < 2; l++)
        {
            R[k][l] = (double)((imgR[k][l] + imgR[k][l + 1]) / 2);
            G[k][l] = (double)((imgG[k][l] + imgG[k][l + 1]) / 2);
            B[k][l] = (double)((imgB[k][l] + imgB[k][l + 1]) / 2);
        }
    }

    for (int k = 0; k < 2; k++)
    {
        for (int l = 0; l < 2; l++)
        {
            outR[k][l] = ceil((double)((R[k][l] + R[k + 1][l]) / 2));
            outG[k][l] = ceil((double)((G[k][l] + G[k + 1][l]) / 2));
            outB[k][l] = ceil((double)((B[k][l] + B[k + 1][l]) / 2));
        }
    }
}

void reducirMatriz9x9a4x4(int imgR[9][9], int imgG[9][9], int imgB[9][9], int outR[4][4], int outG[4][4], int outB[4][4])
{
    double R[9][9];
    double G[9][9];
    double B[9][9];

    int R8x8[8][8];
    int G8x8[8][8];
    int B8x8[8][8];

    for (int k = 0; k < 9; k++)
    {
        for (int l = 0; l < 8; l++)
        {
            R[k][l] = (double)((imgR[k][l] + imgR[k][l + 1]) / 2);
            G[k][l] = (double)((imgG[k][l] + imgG[k][l + 1]) / 2);
            B[k][l] = (double)((imgB[k][l] + imgB[k][l + 1]) / 2);
        }
    }

    for (int k = 0; k < 8; k++)
    {
        for (int l = 0; l < 8; l++)
        {
            R8x8[k][l] = ceil((double)((R[k][l] + R[k + 1][l]) / 2));
            G8x8[k][l] = ceil((double)((G[k][l] + G[k + 1][l]) / 2));
            B8x8[k][l] = ceil((double)((B[k][l] + B[k + 1][l]) / 2));
        }
    }

    algoritmo2Para1080p(R8x8, G8x8, B8x8, outR, outG, outB);
}

void reducirMatriz9x9a2x2(int imgR[9][9], int imgG[9][9], int imgB[9][9], int outR[2][2], int outG[2][2], int outB[2][2])
{
    double R[9][9];
    double G[9][9];
    double B[9][9];

    int R8x8[8][8];
    int G8x8[8][8];
    int B8x8[8][8];

    for (int k = 0; k < 9; k++)
    {
        for (int l = 0; l < 8; l++)
        {
            R[k][l] = (double)((imgR[k][l] + imgR[k][l + 1]) / 2);
            G[k][l] = (double)((imgG[k][l] + imgG[k][l + 1]) / 2);
            B[k][l] = (double)((imgB[k][l] + imgB[k][l + 1]) / 2);
        }
    }

    for (int k = 0; k < 8; k++)
    {
        for (int l = 0; l < 8; l++)
        {
            R8x8[k][l] = ceil((double)((R[k][l] + R[k + 1][l]) / 2));
            G8x8[k][l] = ceil((double)((G[k][l] + G[k + 1][l]) / 2));
            B8x8[k][l] = ceil((double)((B[k][l] + B[k + 1][l]) / 2));
        }
    }

    algoritmo2Para4K(R8x8, G8x8, B8x8, outR, outG, outB);
}

__global__ void reduccion720(int **imgR, int **imgG, int **imgB, int **outR, int **outG, int **outB, int *numeroColumnasImg, int *NUMTHREADS)
{
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;

    if (*NUMTHREADS<=240){
        int filaInicial, filaFinal; //, threadId = *(int *)args;

        int numeroFilasImg = 240; // 720/3
        filaInicial = (numeroFilasImg / *NUMTHREADS) * threadId;
        filaFinal = filaInicial + ((numeroFilasImg / *NUMTHREADS) - 1);

        for (int i = filaInicial; i <= filaFinal; i++)
        {
            for (int j = 0; j < *numeroColumnasImg; j++)
            {
                int R3x3[3][3];
                int G3x3[3][3];
                int B3x3[3][3];

                int indexFilaActual = (i * 3);
                int indexColumnaActual = (j * 3);

                for (int k = 0; k < 3; k++)
                {
                    for (int l = 0; l < 3; l++)
                    {
                        R3x3[k][l] = imgR[indexFilaActual + k][indexColumnaActual + l];
                        G3x3[k][l] = imgG[indexFilaActual + k][indexColumnaActual + l];
                        B3x3[k][l] = imgB[indexFilaActual + k][indexColumnaActual + l];
                    }
                }

                int R2x2[2][2];
                int G2x2[2][2];
                int B2x2[2][2];

                reducirMatriz3x3a2x2(R3x3, G3x3, B3x3, R2x2, G2x2, B2x2);

                int indexFilaActualOUT = (i * 2);
                int indexColumnaActualOUT = (j * 2);

                for (int k = 0; k < 2; k++)
                {
                    for (int l = 0; l < 2; l++)
                    {
                        outR[indexFilaActualOUT + k][indexColumnaActualOUT + l] = R2x2[k][l];
                        outG[indexFilaActualOUT + k][indexColumnaActualOUT + l] = G2x2[k][l];
                        outB[indexFilaActualOUT + k][indexColumnaActualOUT + l] = B2x2[k][l];
                    }
                }
            }
        }
    }else{

    }
}

void *reduccion1080(void *args)
{
    int filaInicial, filaFinal, threadId = *(int *)args;

    int numeroFilasImg = 120; // 1080/9

    filaInicial = (numeroFilasImg / NUMTHREADS) * threadId;
    filaFinal = filaInicial + ((numeroFilasImg / NUMTHREADS) - 1);

    for (int i = filaInicial; i <= filaFinal; i++)
    {
        for (int j = 0; j < numeroColumnasImg; j++)
        {
            int R9x9[9][9];
            int G9x9[9][9];
            int B9x9[9][9];

            int indexFilaActual = (i * 9);
            int indexColumnaActual = (j * 9);

            for (int k = 0; k < 9; k++)
            {
                for (int l = 0; l < 9; l++)
                {
                    R9x9[k][l] = imgR[indexFilaActual + k][indexColumnaActual + l];
                    G9x9[k][l] = imgG[indexFilaActual + k][indexColumnaActual + l];
                    B9x9[k][l] = imgB[indexFilaActual + k][indexColumnaActual + l];
                }
            }

            int R4x4[4][4];
            int G4x4[4][4];
            int B4x4[4][4];

            reducirMatriz9x9a4x4(R9x9, G9x9, B9x9, R4x4, G4x4, B4x4);

            int indexFilaActualOUT = (i * 4);
            int indexColumnaActualOUT = (j * 4);

            for (int k = 0; k < 4; k++)
            {
                for (int l = 0; l < 4; l++)
                {
                    outR[indexFilaActualOUT + k][indexColumnaActualOUT + l] = R4x4[k][l];
                    outG[indexFilaActualOUT + k][indexColumnaActualOUT + l] = G4x4[k][l];
                    outB[indexFilaActualOUT + k][indexColumnaActualOUT + l] = B4x4[k][l];
                }
            }
        }
    }
}

void *reduccion4k(void *args)
{
    int filaInicial, filaFinal, threadId = *(int *)args;

    int numeroFilasImg = 240; // 2160/9
    filaInicial = (numeroFilasImg / NUMTHREADS) * threadId;
    filaFinal = filaInicial + ((numeroFilasImg / NUMTHREADS) - 1);

    for (int i = filaInicial; i <= filaFinal; i++)
    {
        for (int j = 0; j < numeroColumnasImg; j++)
        {
            int R9x9[9][9];
            int G9x9[9][9];
            int B9x9[9][9];

            int indexFilaActual = (i * 9);
            int indexColumnaActual = (j * 9);

            for (int k = 0; k < 9; k++)
            {
                for (int l = 0; l < 9; l++)
                {
                    R9x9[k][l] = imgR[indexFilaActual + k][indexColumnaActual + l];
                    G9x9[k][l] = imgG[indexFilaActual + k][indexColumnaActual + l];
                    B9x9[k][l] = imgB[indexFilaActual + k][indexColumnaActual + l];
                }
            }

            int R2x2[2][2];
            int G2x2[2][2];
            int B2x2[2][2];

            reducirMatriz9x9a2x2(R9x9, G9x9, B9x9, R2x2, G2x2, B2x2);

            int indexFilaActualOUT = (i * 2);
            int indexColumnaActualOUT = (j * 2);

            for (int k = 0; k < 2; k++)
            {
                for (int l = 0; l < 2; l++)
                {
                    outR[indexFilaActualOUT + k][indexColumnaActualOUT + l] = R2x2[k][l];
                    outG[indexFilaActualOUT + k][indexColumnaActualOUT + l] = G2x2[k][l];
                    outB[indexFilaActualOUT + k][indexColumnaActualOUT + l] = B2x2[k][l];
                }
            }
        }
    }
}

int main(int argc, char **argv)
{    
    /* char* nombreEntrada = argv[1];
    char* nombreSalida = argv[2];
    NUMTHREADS = atoi(argv[3]); */

    string nombreEntrada = "imagen720p.jpg";
    string nombreSalida = "imagen720-a480CUDAAAAAAA.jpg";
    NUMTHREADS=2;

    //ofstream file;

    // Leer imágen
    Mat img = imread(nombreEntrada, CV_LOAD_IMAGE_COLOR);

    //cout << "La imagen tiene " << img.rows << " pixeles de alto x "<< img.cols << " pixeles de ancho" << endl;

    // Inicio Correción tamaño (si necesita)
    int nuevoNcolumnas = 0;

    if (img.rows == 720)
    {
        if (img.cols % 3 != 0)
        {
            nuevoNcolumnas = (img.cols / 3) * 3;
            img = cambiarTamanoImagen(img, img.rows, nuevoNcolumnas);
        }
    }
    else if (img.rows == 1080)
    {
        if (img.cols % 9 != 0)
        {
            nuevoNcolumnas = (img.cols / 9) * 9;
            img = cambiarTamanoImagen(img, img.rows, nuevoNcolumnas);
        }
    }
    else if (img.rows == 2160)
    {
        if (img.cols % 9 != 0)
        {
            nuevoNcolumnas = (img.cols / 9) * 9;
            img = cambiarTamanoImagen(img, img.rows, nuevoNcolumnas);
        }
    }
    else
    {
        cout << "Resolución no permitida" << endl;
    }
    // Fin Correción tamaño (si necesita)

    // cout << "La imagen tiene " << img.rows << " pixeles de alto x " << img.cols << " pixeles de ancho" << endl;

    // Comienzo creación de matrices
    const int rows = img.rows;
    const int cols = img.cols;

    imgR = new int *[rows];
    for (size_t i = 0; i < rows; ++i)
        imgR[i] = new int[cols];

    imgG = new int *[rows];
    for (size_t i = 0; i < rows; ++i)
        imgG[i] = new int[cols];

    imgB = new int *[rows];
    for (size_t i = 0; i < rows; ++i)
        imgB[i] = new int[cols];

    for (int i = 0; i < img.rows; i++)
    {
        for (int j = 0; j < img.cols; j++)
        {
            imgR[i][j] = img.at<cv::Vec3b>(i, j)[2];
            imgG[i][j] = img.at<cv::Vec3b>(i, j)[1];
            imgB[i][j] = img.at<cv::Vec3b>(i, j)[0];
        }
    }

    const int outRows = 480;
    const int outCols = (outRows * img.cols) / img.rows;

    // Output image
    Mat imgOut(outRows, outCols, CV_8UC3);

    outR = new int *[outRows];
    for (size_t i = 0; i < outRows; ++i)
        outR[i] = new int[outCols];

    outG = new int *[outRows];
    for (size_t i = 0; i < outRows; ++i)
        outG[i] = new int[outCols];

    outB = new int *[outRows];
    for (size_t i = 0; i < outRows; ++i)
        outB[i] = new int[outCols];

    // Fin creación de matrices

    //************************** CUDA **********************************

    int **d_imgR;
    int **d_imgG;
    int **d_imgB;
    int **d_outR;
    int **d_outG;
    int **d_outB;
    int *d_numeroColumnasImg;
    int *d_NUMTHREADS;
    

    int sizeIn = sizeof(imgR); // Size sirve para todas las img
    int sizeOut = sizeof(outR); // Size sirve para todas las out
    int sizeNumeroColumnasImg = sizeof(numeroColumnasImg); 
    int sizeNUMTHREADS = sizeof(NUMTHREADS); 

    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_imgR, sizeIn);
    hipMalloc((void **)&d_imgG, sizeIn);
    hipMalloc((void **)&d_imgB, sizeIn);

    hipMalloc((void **)&d_outR, sizeOut);
    hipMalloc((void **)&d_outG, sizeOut);
    hipMalloc((void **)&d_outB, sizeOut);

    hipMalloc((void **)&d_numeroColumnasImg, sizeNumeroColumnasImg);
    hipMalloc((void **)&d_NUMTHREADS, sizeNUMTHREADS);


    numeroColumnasImg = cols / 3;


    // Copy inputs to device
    hipMemcpy(d_imgR, imgR, sizeIn, hipMemcpyHostToDevice);
    hipMemcpy(d_imgG, imgG, sizeIn, hipMemcpyHostToDevice);
    hipMemcpy(d_imgB, imgB, sizeIn, hipMemcpyHostToDevice);
    hipMemcpy(d_numeroColumnasImg, numeroColumnasImg, sizeNumeroColumnasImg, hipMemcpyHostToDevice);
    hipMemcpy(d_NUMTHREADS, NUMTHREADS, sizeNUMTHREADS, hipMemcpyHostToDevice);

    int NUMTHREADSPerBlock = NUMTHREADS/BLOCKSPERGRID;
    // Launch add() kernel on GPU with N blocks
    reduccion720<<<BLOCKSPERGRID, NUMTHREADSPerBlock>>>(d_imgR, d_imgG, d_imgB, d_outR, d_outG, d_outB, d_numeroColumnasImg, d_NUMTHREADS);

    // Copy result back to host
    hipMemcpy(outR, d_outR, sizeOut, hipMemcpyDeviceToHost);
    hipMemcpy(outG, d_outG, sizeOut, hipMemcpyDeviceToHost);
    hipMemcpy(outB, d_outB, sizeOut, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_imgR); hipFree(d_imgG); hipFree(d_imgB);hipFree(d_outR); hipFree(d_outG); hipFree(d_outB); hipFree(d_numeroColumnasImg); hipFree(d_NUMTHREADS);


    //************************** CUDA **********************************


    /*

    //Inicio Conversión**********************************
    int numeroFilasImg = 0;

    //  Creación hilos y empezar toma de tiempo
    int threadId[NUMTHREADS], i, *retval;
    pthread_t thread[NUMTHREADS];

    struct timeval tval_before, tval_after, tval_result;
    gettimeofday(&tval_before, NULL);
    //  **************

    if (rows == 720)
    {
        numeroColumnasImg = cols / 3;

        for (i = 0; i < NUMTHREADS; i++)
        {
            threadId[i] = i;
            pthread_create(&thread[i], NULL, reduccion720, &threadId[i]);
        }
    }
    else if (rows == 1080)
    {

        numeroColumnasImg = cols / 9;

        for (i = 0; i < NUMTHREADS; i++)
        {
            threadId[i] = i;
            pthread_create(&thread[i], NULL, reduccion1080, &threadId[i]);
        }
    }
    else if (rows == 2160)
    {

        numeroColumnasImg = cols / 9;

        for (i = 0; i < NUMTHREADS; i++)
        {
            threadId[i] = i;
            pthread_create(&thread[i], NULL, reduccion4k, &threadId[i]);
        }
    }
    else
    {
        cout << "Resolución no permitida" << endl;
    }
    //Fin Conversión*******************

    //Recolección Hilos y finalización toma de tiempo
    for (i = 0; i < NUMTHREADS; i++)
    {
        pthread_join(thread[i], (void **)&retval);
    }

    gettimeofday(&tval_after, NULL);
    timersub(&tval_after, &tval_before, &tval_result);

    

    if (rows == 720){
        ofstream file;
        file.open("./720.txt", ofstream::app);
        file << NUMTHREADS << " HILOS: " << (long double)tval_result.tv_sec + (long double)(tval_result.tv_usec)/1000000 << endl;
        file.close();
    }else if (rows == 1080){
        ofstream file;
        file.open("./1080.txt", ofstream::app);
        file << NUMTHREADS << " HILOS: " << (long double)tval_result.tv_sec + (long double)(tval_result.tv_usec)/1000000 << endl;
        file.close();
    }else{
        ofstream file;
        file.open("./4k.txt", ofstream::app);
        file << NUMTHREADS << " HILOS: " << (long double)tval_result.tv_sec + (long double)(tval_result.tv_usec)/1000000 << endl;
        file.close();
    }
    //Fin Recolección Hilos y finalización toma de tiempo*****
    */

    //Pasar matrices resultantes a Imagen de salida
    for (int i = 0; i < outRows; i++)
    {
        for (int j = 0; j < outCols; j++)
        {
            imgOut.at<cv::Vec3b>(i, j)[0] = outB[i][j];
            imgOut.at<cv::Vec3b>(i, j)[1] = outG[i][j];
            imgOut.at<cv::Vec3b>(i, j)[2] = outR[i][j];
        }
    }
    //Fin Pasar matrices resultantes a Imagen de salida

    //Inicio borrar matrices
    for (size_t i = 0; i < rows; ++i)
        delete imgR[i];
    delete imgR;

    for (size_t i = 0; i < rows; ++i)
        delete imgG[i];
    delete imgG;

    for (size_t i = 0; i < rows; ++i)
        delete imgB[i];
    delete imgB;

    for (size_t i = 0; i < outRows; ++i)
        delete outR[i];
    delete outR;

    for (size_t i = 0; i < outRows; ++i)
        delete outG[i];
    delete outG;

    for (size_t i = 0; i < outRows; ++i)
        delete outB[i];
    delete outB;

    //Fin Borrar matrices

    // Imprimir Imagen original y convertida. DESCOMENTAR LAS SIGUIENTES LINEAS SI SE QUIEREN VER LAS IMAGENES DE ENTRADA Y SALIDA RESPECTIVAMENTE
    //imshow(nombreEntrada, img);
    //imshow(nombreSalida, imgOut);

    //Guarda imágen de salida en directorio local
    imwrite(nombreSalida, imgOut);
    waitKey(0);

    return 1;
}
